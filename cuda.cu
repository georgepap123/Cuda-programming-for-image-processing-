#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <jpeglib.h>

__global__ void remove_red_green(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        output[index * 3] = 0;
        output[index * 3 + 1] = 0;
        output[index * 3 + 2] = input[index * 3 + 2];
    }
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: %s input_file output_file\n", argv[0]);
        return 1;
    }

    // Open input file
    FILE *input_file = fopen(argv[1], "rb");
    if (!input_file) {
        printf("Error: Failed to open input file\n");
        return 1;
    }

    // Read JPEG header
    struct jpeg_decompress_struct cinfo;
    struct jpeg_error_mgr jerr;
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_decompress(&cinfo);
    jpeg_stdio_src(&cinfo, input_file);
    jpeg_read_header(&cinfo, TRUE);

    // Allocate memory for image data
    int width = cinfo.image_width;
    int height = cinfo.image_height;
    int size = width * height * 3;
    unsigned char *input = (unsigned char *)malloc(size);
    unsigned char *output = (unsigned char *)malloc(size);

    // Read image data
    jpeg_start_decompress(&cinfo);
    while (cinfo.output_scanline < cinfo.output_height) {
        unsigned char *buffer[1];
        buffer[0] = input + cinfo.output_scanline * width * 3;
        jpeg_read_scanlines(&cinfo, buffer, 1);
    }
    jpeg_finish_decompress(&cinfo);
    jpeg_destroy_decompress(&cinfo);
    fclose(input_file);

    // Allocate memory on GPU
    unsigned char *input_gpu, *output_gpu;
    hipMalloc(&input_gpu, size);
    hipMalloc(&output_gpu, size);

    // Copy data to GPU
    hipMemcpy(input_gpu, input, size, hipMemcpyHostToDevice);

    // Launch kernel
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
    remove_red_green<<<grid, block>>>(input_gpu, output_gpu, width, height);
   // Copy data back from GPU
hipMemcpy(output, output_gpu, size, hipMemcpyDeviceToHost);

// Open output file
FILE *output_file = fopen(argv[2], "wb");
if (!output_file) {
    printf("Error: Failed to open output file\n");
    return 1;
}

// Write JPEG header
struct jpeg_compress_struct cinfo_out;
struct jpeg_error_mgr jerr_out;
cinfo_out.err = jpeg_std_error(&jerr_out);
jpeg_create_compress(&cinfo_out);
jpeg_stdio_dest(&cinfo_out, output_file);
cinfo_out.image_width = width;
cinfo_out.image_height = height;
cinfo_out.input_components = 3;
cinfo_out.in_color_space = JCS_RGB;
jpeg_set_defaults(&cinfo_out);
jpeg_set_quality(&cinfo_out, 100, TRUE);
jpeg_start_compress(&cinfo_out, TRUE);

// Write image data
while (cinfo_out.next_scanline < cinfo_out.image_height) {

     unsigned char *buffer[1];
    buffer[0] = output + cinfo_out.next_scanline * width * 3;
    jpeg_write_scanlines(&cinfo_out, buffer, 1);
}
jpeg_finish_compress(&cinfo_out);
jpeg_destroy_compress(&cinfo_out);
fclose(output_file);

// Free memory
free(input);
free(output);
hipFree(input_gpu);
hipFree(output_gpu);

return 0;
}